#include "hip/hip_runtime.h"
#include "lbm3d/core.h"
#include "lbm3d/lagrange_3D.h"

// ball in 3D
// IBM-LBM
// This simulation is used for regression tests of IBM matrix construction
// This simulation is based on sim_IBM2.cu

template < typename TRAITS >
struct MacroLocal : D3Q27_MACRO_Base< TRAITS >
{
	using dreal = typename TRAITS::dreal;
	using idx = typename TRAITS::idx;

	enum { e_fx, e_fy, e_fz, e_vx, e_vy, e_vz, e_rho, N };

	template < typename LBM_DATA, typename LBM_KS >
	CUDA_HOSTDEV static void outputMacro(LBM_DATA &SD, LBM_KS &KS, idx x, idx y, idx z)
	{
		SD.macro(e_rho, x, y, z) = KS.rho;
		SD.macro(e_vx, x, y, z)  = KS.vx;
		SD.macro(e_vy, x, y, z)  = KS.vy;
		SD.macro(e_vz, x, y, z)  = KS.vz;
	}

	template < typename LBM_DATA >
	CUDA_HOSTDEV static void zeroForces(LBM_DATA &SD, idx x, idx y, idx z)
	{
		SD.macro(e_fx, x, y, z) = 0;
		SD.macro(e_fy, x, y, z) = 0;
		SD.macro(e_fz, x, y, z) = 0;
	}

	template < typename LBM_DATA, typename LBM_KS >
	CUDA_HOSTDEV static void copyQuantities(LBM_DATA &SD, LBM_KS &KS, idx x, idx y, idx z)
	{
		KS.lbmViscosity = SD.lbmViscosity;
		KS.fx = SD.macro(e_fx, x, y, z);
		KS.fy = SD.macro(e_fy, x, y, z);
		KS.fz = SD.macro(e_fz, x, y, z);
	}
};

template < typename NSE >
struct StateLocal : State<NSE>
{
	using TRAITS = typename NSE::TRAITS;
	using BC = typename NSE::BC;
	using MACRO = typename NSE::MACRO;
	using BLOCK = LBM_BLOCK< NSE >;

	using State<NSE>::nse;
	using State<NSE>::ibm;
	using State<NSE>::vtk_helper;
	using State<NSE>::id;

	using idx = typename TRAITS::idx;
	using real = typename TRAITS::real;
	using dreal = typename TRAITS::dreal;
	using point_t = typename TRAITS::point_t;
	using lat_t = Lattice<3, real, idx>;

	dreal lbm_inflow_vx = 0;
	bool firstrun=true;
	real ball_diameter=0.01;
	real ball_c[3];

	virtual bool outputData(const BLOCK& block, int index, int dof, char *desc, idx x, idx y, idx z, real &value, int &dofs)
	{
		int k=0;
		if (index==k++)
		{
			switch (dof)
			{
				case 0: return vtk_helper("lbm_velocity", block.hmacro(MACRO::e_vx,x,y,z), 3, desc, value, dofs);
				case 1: return vtk_helper("lbm_velocity", block.hmacro(MACRO::e_vy,x,y,z), 3, desc, value, dofs);
				case 2: return vtk_helper("lbm_velocity", block.hmacro(MACRO::e_vz,x,y,z), 3, desc, value, dofs);
			}
		}
		if (index==k++)
		{
			switch (dof)
			{
				case 0: return vtk_helper("lbm_force", block.hmacro(MACRO::e_fx,x,y,z), 3, desc, value, dofs);
				case 1: return vtk_helper("lbm_force", block.hmacro(MACRO::e_fy,x,y,z), 3, desc, value, dofs);
				case 2: return vtk_helper("lbm_force", block.hmacro(MACRO::e_fz,x,y,z), 3, desc, value, dofs);
			}
		}
		if (index==k++) return vtk_helper("lbm_density", block.hmacro(MACRO::e_rho,x,y,z), 1, desc, value, dofs);
		if (index==k++) return vtk_helper("lbm_density_fluctuation", block.hmacro(MACRO::e_rho,x,y,z)-1.0, 1, desc, value, dofs);
		if (index==k++)
		{
			switch (dof)
			{
				case 0: return vtk_helper("velocity", nse.lat.lbm2physVelocity(block.hmacro(MACRO::e_vx,x,y,z)), 3, desc, value, dofs);
				case 1: return vtk_helper("velocity", nse.lat.lbm2physVelocity(block.hmacro(MACRO::e_vy,x,y,z)), 3, desc, value, dofs);
				case 2: return vtk_helper("velocity", nse.lat.lbm2physVelocity(block.hmacro(MACRO::e_vz,x,y,z)), 3, desc, value, dofs);
			}
		}
		if (index==k++)
		{
			switch (dof)
			{
				case 0: return vtk_helper("force", nse.lat.lbm2physForce(block.hmacro(MACRO::e_fx,x,y,z)), 3, desc, value, dofs);
				case 1: return vtk_helper("force", nse.lat.lbm2physForce(block.hmacro(MACRO::e_fy,x,y,z)), 3, desc, value, dofs);
				case 2: return vtk_helper("force", nse.lat.lbm2physForce(block.hmacro(MACRO::e_fz,x,y,z)), 3, desc, value, dofs);
			}
		}
		//if (index==k++) return vtk_helper("density", block.hmacro(MACRO::e_rho,x,y,z)*nse.physFluidDensity, 1, desc, value, dofs);
		return false;
	}

	virtual void updateKernelVelocities()
	{
		for (auto& block : nse.blocks)
		{
			block.data.inflow_rho = 1;
			block.data.inflow_vx = lbm_inflow_vx;
			block.data.inflow_vy = 0;
			block.data.inflow_vz = 0;
		}
	}

	virtual void setupBoundaries()
	{
		nse.setBoundaryX(0, BC::GEO_INFLOW); // left
		nse.setBoundaryX(nse.lat.global.x()-1, BC::GEO_OUTFLOW_EQ);// right
		nse.setBoundaryY(0, BC::GEO_INFLOW); // back
		nse.setBoundaryY(nse.lat.global.y()-1, BC::GEO_INFLOW);// front
		nse.setBoundaryZ(0, BC::GEO_INFLOW);// top
		nse.setBoundaryZ(nse.lat.global.z()-1, BC::GEO_INFLOW);// bottom
	}

	StateLocal(const std::string& id, const TNL::MPI::Comm& communicator, lat_t lat)
		: State<NSE>(id, communicator, lat)
	{}
};

// ball discretization algorithm: https://www.cmu.edu/biolphys/deserno/pdf/sphere_equi.pdf
template < typename STATE >
void drawFixedSphere(STATE &state, double cx, double cy, double cz, double radius, double sigma)
{
	using real = typename STATE::TRAITS::real;
	using point_t = typename STATE::TRAITS::point_t;

	// based on sigma, estimate N
	real surface = 4.0*PI*radius*radius;
	// sigma is diagonal of a "quasi-square" that 4 points on the sphere surface form
	// so the quasi-square has area = b^2 where b^2 + b^2 = sigma^2, i.e., b^2 = 1/2*sigma^2
	real wanted_unit_area = sigma*sigma / 2.0;
	// count how many of these
	real count = surface/wanted_unit_area;
	int N = ceil(count);

	int points=0;
//	real a = 4.0*PI*radius*radius/(real)N;
	real a = 4.0*PI/(real)N;
	real d = sqrt(a);
//	int Mtheta = (int)(PI/d);
	int Mtheta = floor(PI/d);
	real dtheta = PI/Mtheta;
	real dphi = a/dtheta;
	for (int m = 0; m < Mtheta; m++)
	{
		// for a given phi and theta:
		real theta = PI*(m+0.5)/(real)Mtheta;
//		int Mphi = (int)(2.0*PI*sin(theta)/dphi);
		int Mphi = floor(2.0*PI*sin(theta)/dphi);
		for (int n = 0; n<Mphi; n++)
		{
			real phi = 2.0*PI*n/(real)Mphi;
			point_t fp;
			fp.x() = cx + radius * cos( phi ) * sin( theta );
			fp.y() = cy + radius * sin( phi ) * sin( theta );
			fp.z() = cz + radius * cos( theta );
			state.ibm.LL.push_back(fp);
			points++;
		}
	}
	spdlog::info("added {} lagrangian points", points);

	real sigma_min = state.ibm.computeMinDist();
	real sigma_max = state.ibm.computeMaxDistFromMinDist(sigma_min);

	spdlog::info("Ball surface: wanted sigma {:e} ({:f} i.e. {:d} points), wanted_unit_area {:e}, sigma_min {:e}, sigma_max {:e}", sigma, count, N, wanted_unit_area, sigma_min, sigma_max);
//	spdlog::info("Added {} Lagrangian points (requested {}) partial area {:e}",points, N, a);
//	spdlog::info("Lagrange created: WuShuCompute {} ws_regularDirac {}", state.ibm.WuShuCompute, (state.ibm.ws_regularDirac)?"true":"false");
	spdlog::info("h=physdl {:e} sigma min {:e} sigma_ku_h {:e}", state.nse.lat.physDl, sigma_min, sigma_min/state.nse.lat.physDl);
	spdlog::info("h=physdl {:e} sigma max {:e} sigma_ku_h {:e}", state.nse.lat.physDl, sigma_max, sigma_max/state.nse.lat.physDl);

	state.writeVTK_Points("ball.vtk",0,0,state.ibm);
}

template < typename NSE >
int sim(int RES=2, double i_Re=1000, double nasobek=2.0, int dirac_delta=2, int method=0, int compute=5)
{
	using idx = typename NSE::TRAITS::idx;
	using real = typename NSE::TRAITS::real;
	using point_t = typename NSE::TRAITS::point_t;
	using lat_t = Lattice<3, real, idx>;

	int block_size=32;
	real BALL_DIAMETER = 0.01;
	real real_domain_height= BALL_DIAMETER*11;// [m]
	//real real_domain_length= BALL_DIAMETER*11;// [m] // extra 1cm on both sides
	idx LBM_Y = RES*block_size; // for 4 cm
	idx LBM_Z = LBM_Y;
	real PHYS_DL = real_domain_height/((real)LBM_Y);
	idx LBM_X = LBM_Y;//(int)(real_domain_length/PHYS_DL)+2;//block_size;//16*RESOLUTION;
	point_t PHYS_ORIGIN = {0., 0., 0.};

	// zvolit Re + LBM VELOCITY + PHYS_VISCOSITY
//	real i_Re = ;
	real i_LBM_VELOCITY = 0.07; // Geier
	real i_PHYS_VISCOSITY = 0.00001; // proc ne?
	// mam:
	real i_LBM_VISCOSITY = i_LBM_VELOCITY * BALL_DIAMETER / PHYS_DL / i_Re;
	real i_PHYS_VELOCITY = i_PHYS_VISCOSITY * i_Re / BALL_DIAMETER;
	fmt::print("input phys velocity {:f}\ninput lbm velocity {:f}\nRe {:f}\nlbm viscosity{:f}\nphys viscosity {:f}\n", i_PHYS_VELOCITY, i_LBM_VELOCITY, i_Re, i_LBM_VISCOSITY, i_PHYS_VISCOSITY);

	real LBM_VISCOSITY = i_LBM_VISCOSITY;// 0.0001*RES;//*SIT;//1.0/6.0; /// GIVEN: optimal is 1/6
	real PHYS_VISCOSITY = i_PHYS_VISCOSITY;//0.00001;// [m^2/s] fluid viscosity of water
	real Re=i_Re;//200;

//	real INIT_TIME = 1.0; // [s]
	real PHYS_DT = LBM_VISCOSITY / PHYS_VISCOSITY*PHYS_DL*PHYS_DL;

	// initialize the lattice
	lat_t lat;
	lat.global = typename lat_t::CoordinatesType( LBM_X, LBM_Y, LBM_Z );
	lat.physOrigin = PHYS_ORIGIN;
	lat.physDl = PHYS_DL;
	lat.physDt = PHYS_DT;
	lat.physViscosity = PHYS_VISCOSITY;

	const std::string state_id = fmt::format("sim_IBM3_{}_{}_dirac_{}_res_{}_Re_{}_nas_{:05.4f}_compute_{}", NSE::COLL::id, (method>0)?"original":"modified", dirac_delta, RES, Re, nasobek, compute);
	StateLocal<NSE> state(state_id, MPI_COMM_WORLD, lat);

	//if (state.isMark())
	//	return 0;

	state.lbm_inflow_vx = i_LBM_VELOCITY;
	state.nse.physCharLength = BALL_DIAMETER; // [m]
	state.ball_diameter = BALL_DIAMETER; // [m]
	//state.nse.physFluidDensity = 1000.0; // [kg/m^3]

	state.cnt[PRINT].period = 0.1;
	state.nse.physFinalTime = 0.0;

//	state.cnt[VTK3D].period = 1.0;
	state.cnt[VTK2D].period = 1.0;
	state.cnt[VTK1D].period = 1.0;

	// select compute method
	IbmCompute computeVariant;
	switch (compute)
	{
		case 0: computeVariant = IbmCompute::GPU; break;
		case 1: computeVariant = IbmCompute::CPU; break;
		case 2: computeVariant = IbmCompute::Hybrid; break;
		case 3: computeVariant = IbmCompute::Hybrid_zerocopy; break;
		default:
			spdlog::warn("Unknown parameter compute={}, selecting GPU as the default.", compute);
			computeVariant = IbmCompute::GPU;
			break;
	}

	// add cuts
	state.add2Dcut_X(LBM_X/2,"cut_X");
//	state.add2Dcut_X(2*BALL_DIAMETER/PHYS_DL,"cut_Xball");
	state.add2Dcut_Y(LBM_Y/2,"cut_Y");
	state.add2Dcut_Z(LBM_Z/2,"cut_Z");

	state.ball_c[0] = 2*state.ball_diameter;
	state.ball_c[1] = 5.5*state.ball_diameter;
	state.ball_c[2] = 5.5*state.ball_diameter;
	// create a filament
	real sigma = nasobek * PHYS_DL;
	drawFixedSphere(state, state.ball_c[0], state.ball_c[1], state.ball_c[2], state.ball_diameter/2.0, sigma);

	// configure IBM
	state.ibm.computeVariant = computeVariant;
	state.ibm.diracDeltaTypeEL = dirac_delta;
	if (method == 0)
		state.ibm.methodVariant = IbmMethod::modified;
	else
		state.ibm.methodVariant = IbmMethod::original;
	state.ibm.mtx_output = true;

	execute(state);

	return 0;
}

template < typename TRAITS=TraitsSP >
void run(int res, double Re, double h, int dirac, int method, int compute)
{
	using COLL = D3Q27_CUM<TRAITS>;
	using NSE_CONFIG = LBM_CONFIG<
				TRAITS,
				D3Q27_KernelStruct,
				NSE_Data_ConstInflow< TRAITS >,
				COLL,
				typename COLL::EQ,
				D3Q27_STREAMING< TRAITS >,
				D3Q27_BC_All,
				MacroLocal< TRAITS >,
				D3Q27_MACRO_Void< TRAITS >
			>;

	sim<NSE_CONFIG>(res, Re, h, dirac, method, compute);
}

int main(int argc, char **argv)
{
	TNLMPI_INIT mpi(argc, argv);

	// here "h" is the Lagrangian/Eulerian spacing ratio
//	const double hvals[] = { 2.0, 1.5, 1.0, 0.75, 0.50, 0.25 };
//	const double hvals[] = { 2.0 };
//	const double hvals[] = { 0.125 };
	const double hvals[] = { 0.25, 0.5, 0.75, 1.0, 1.5, 2.0 };
//	const double hvals[] = { 2.0, 1.8, 1.6, 1.4, 1.2, 1.0, 0.9, 0.8, 0.75, 0.50 };
//	const double hvals[] = { 2.0, 1.8, 1.6, 1.4, 1.2, 1.0, 0.75, 0.50, 0.25, 0.125 };
//	const double hvals[] = { 3.0, 2.5, 2.0, 1.8, 1.6, 1.4, 1.2, 1.0, 0.9, 0.8, 0.7, 0.6, 0.5, 0.4, 0.3, 0.2, 0.1 }; // ostrava
	int hmax = sizeof(hvals)/sizeof(double);
	double h=1.0;

	bool use_command_line=true;

	if (!use_command_line)
	{
		int dirac=1;
		int res=2; // 3, 6, 12
		int hi=3;
		int method=0; //0 = modified
		int Re=100;
		int compute=5;
//		for (int Re=100;Re<=200; Re+=100)
//		for (hi=0;hi<hmax;hi++)
//		for (method=0;method<=1;method++)
//		for (res=3;res<=5;res++)
//		for (dirac=1;dirac<=4;dirac++)
//		for (compute=1;compute<=6;compute++)
		{
			if (hi<hmax) h=hvals[hi];
			run(res, (double)Re, h, dirac, method, compute);
		}
	} else
	{
		const int pars=6;
		if (argc <= pars)
		{
			fprintf(stderr, "error: %d parameters required:\n %s method{0,1} dirac{1,2,3,4} Re{100,200} hi[0,%d] res[1,22] compute[1,7]\n", pars, argv[0],hmax-1);
			return 1;
		}
		int method = atoi(argv[1]);	// 0=modified 1=original
		int dirac = atoi(argv[2]);
		int Re = atoi(argv[3]);		// type=0,1,2 (geometry selection)
		int hi = atoi(argv[4]);		// index in the hvals
		int res = atoi(argv[5]);	// res=1,2,3
		int compute = atoi(argv[6]); // compute=0,1,2,3

		if (method > 1 || method < 0) { fprintf(stderr, "error: method=%d out of bounds [0, 1]\n",method); return 1; }
		if (dirac < 1 || dirac > 4) { fprintf(stderr, "error: dirac=%d out of bounds [1,4]\n",dirac); return 1; }
		if (hi >= hmax || hi < 0) { fprintf(stderr, "error: hi=%d out of bounds [0, %d]\n",hi,hmax-1); return 1; }
		if (res < 1) { fprintf(stderr, "error: res=%d out of bounds [1, ...]\n",res); return 1; }
		if (compute < 0 || compute > 3) { fprintf(stderr, "error: compute=%d out of bounds [0,3]\n",compute); return 1; }
		if (hi<hmax) h=hvals[hi];
		run(res, (double)Re, h, dirac, method, compute);
	}
	return 0;
}
